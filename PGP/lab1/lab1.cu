
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(double *arr, double *arr2, unsigned long n) {
	unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned long offset = blockDim.x * gridDim.x;
//	printf("%d %d %d\n", blockIdx.x, threadIdx.x, idx);
	while (idx < n) {
		arr[idx] += arr2[idx];
		idx += offset;
	}
}

int main() {
	unsigned long i, n;
	scanf("%lu", &n);
	double *arr = (double *)malloc(sizeof(double) * n);
	double *arr2 = (double *)malloc(sizeof(double) * n);
	for (i = 0; i < n; i++)
		scanf("%lf", &arr[i]);
	for (i = 0; i < n; i++)
		scanf("%lf", &arr2[i]);
	double *dev_arr;
	double *dev_arr2;
	hipMalloc(&dev_arr, sizeof(double) * n);
	hipMalloc(&dev_arr2, sizeof(double) * n);
	hipMemcpy(dev_arr, arr, sizeof(double) * n, hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr2, sizeof(double) * n, hipMemcpyHostToDevice);

	kernel<<<32, 32>>>(dev_arr, dev_arr2, n);

	hipMemcpy(arr, dev_arr, sizeof(double) * n, hipMemcpyDeviceToHost);
	hipMemcpy(arr2, dev_arr2, sizeof(double) * n, hipMemcpyDeviceToHost);
	for(i = 0; i < n; i++)
		printf("%.10e ", arr[i]);
	printf("\n");

	hipFree(dev_arr);
	hipFree(dev_arr2);
	free(arr);
	free(arr2);
	return 0;
}